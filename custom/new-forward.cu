#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include "gpu-new-forward.h"
#include <mma.h>
using namespace nvcuda;

// do padding for the dimensions
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;
#define MATRIX_M 16
#define MATRIX_N 912
#define MATRIX_K 208

#define errCheck(ans) { checkError((ans), __FILE__, __LINE__); }
inline void checkError(hipError_t err, const char * file, int line, bool abort = true) {
    if (err != hipSuccess) {
        fprintf(stderr, "GPU Error: %s --> %s:%d\n", hipGetErrorString(err), file, line);
        if (abort) exit(err);
    }
}

#define TILE_WIDTH_16 16
//#define TILE_WIDTH_32 32
__global__ void conv_forward_kernel_16(float *output, __half *input, __half *mask,
     const int Batch, const int Map_out, const int Channel,
     const int Height, const int Width, const int K)
{
    //__shared__ __half tileA[TILE_WIDTH_16][TILE_WIDTH_16];
    //__shared__ __half tileB[TILE_WIDTH_16][TILE_WIDTH_16];

    float alpha = 1.0;
    float beta=0.0;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    int b = blockIdx.z;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH_16 + ty;
    int column = blockIdx.x * TILE_WIDTH_16 + tx;

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int numMatAColumns = Channel*K*K;
    __half acc=0.0;

    int numIterations = ceil((numMatAColumns*1.0)/TILE_WIDTH_16);

    #define out_4d(i3, i2, i1, i0) &output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) &input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) &mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    int lda = Map_out;
    int ldb = Channel*K*K;
    int ldc = Map_out;

    for(int i=0; i<numIterations; i++){
        int tempCol = i*TILE_WIDTH_16+tx;
        int tempRow = i*TILE_WIDTH_16+ty;

        int W_m = row;
        int W_c = tempCol/(K*K);
        int W_h = (tempCol%(K*K))/K;
        int W_w = (tempCol%(K*K))%K;

        if((tempCol < numMatAColumns) && (row < Map_out) )
            wmma::load_matrix_sync(a_frag, mask_4d(W_m, W_c, W_h, W_w), lda);

        int X_b = b;
        int X_c = tempRow/(K*K);
        int X_p = (tempRow%(K*K))/K;
        int X_q = (tempRow%(K*K))%K;
        int X_h = column / Width_out;
        int X_w = column % Width_out;

        if((tempRow < numMatAColumns) && (column < Height_out*Width_out) )
            wmma::load_matrix_sync(b_frag, in_4d(X_b, X_c, X_p + X_h, X_q + X_w), ldb);

        // Perform the matrix multiplication
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    int Y_b = b;
    int Y_m = row;
    int Y_h = column / Width_out;
    int Y_w = column % Width_out;

    if ((row < Map_out) && (column < Height_out*Width_out)) {
        wmma::load_matrix_sync(c_frag, out_4d(Y_b, Y_m, Y_h, Y_w), ldc, wmma::mem_row_major);

        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(out_4d(Y_b, Y_m, Y_h, Y_w), c_frag, ldc, wmma::mem_row_major);
   }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}
/*
 __global__ void half_to_float(__half *in_array, float *out,  int outlen, int M)
{
    const int map = threadIdx.y + blockDim.y*blockIdx.y;
    const int z = blockIdx.z;
    const int houtwout = threadIdx.x + blockDim.x*blockIdx.x;
    if(map < M && houtwout < outlen) out[z*outlen*M + map*outlen+houtwout] = __half2float(in_array[z*outlen*M + map*outlen+houtwout]);
}

__global__ void conv_forward_kernel_32(__half *output, __half *input, __half *mask,
     const int Batch, const int Map_out, const int Channel,
     const int Height, const int Width, const int K)
{
    __shared__ __half tileA[TILE_WIDTH_32][TILE_WIDTH_32];
    __shared__ __half tileB[TILE_WIDTH_32][TILE_WIDTH_32];

    int b = blockIdx.z;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH_32 + ty;
    int column = blockIdx.x * TILE_WIDTH_32 + tx;

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int numMatAColumns = Channel*K*K;
    __half acc=0.0;

    int numIterations = ceil((numMatAColumns*1.0)/TILE_WIDTH_32);

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    for(int i=0; i<numIterations; i++){
        int tempCol = i*TILE_WIDTH_32+tx;
        int tempRow = i*TILE_WIDTH_32+ty;
        tileA[ty][tx] = 0.0;
        tileB[ty][tx] = 0.0;

        int W_m = row;
        int W_c = tempCol/(K*K);
        int W_h = (tempCol%(K*K))/K;
        int W_w = (tempCol%(K*K))%K;

        if((tempCol < numMatAColumns) && (row < Map_out) )
            tileA[ty][tx] = mask_4d(W_m, W_c, W_h, W_w);
        else tileA[ty][tx] = 0.0;

        int X_b = b;
        int X_c = tempRow/(K*K);
        int X_p = (tempRow%(K*K))/K;
        int X_q = (tempRow%(K*K))%K;
        int X_h = column / Width_out;
        int X_w = column % Width_out;

        if((tempRow < numMatAColumns) && (column < Height_out*Width_out) )
            tileB[ty][tx] = in_4d(X_b, X_c, X_p + X_h, X_q + X_w);
        else tileB[ty][tx] = 0.0;

        __syncthreads();

        acc  += tileA[ty][0] * tileB[0][tx]
        +tileA[ty][1] * tileB[1][tx]
        +tileA[ty][2] * tileB[2][tx]
        +tileA[ty][3] * tileB[3][tx]
        +tileA[ty][4] * tileB[4][tx]
        +tileA[ty][5] * tileB[5][tx]
        +tileA[ty][6] * tileB[6][tx]
        +tileA[ty][7] * tileB[7][tx]
        +tileA[ty][8] * tileB[8][tx]
        +tileA[ty][9] * tileB[9][tx]
        +tileA[ty][10] * tileB[10][tx]
        +tileA[ty][11] * tileB[11][tx]
        +tileA[ty][12] * tileB[12][tx]
        +tileA[ty][13] * tileB[13][tx]
        +tileA[ty][14] * tileB[14][tx]
        +tileA[ty][15] * tileB[15][tx]
        +tileA[ty][16] * tileB[16][tx]
        +tileA[ty][17] * tileB[17][tx]
        +tileA[ty][18] * tileB[18][tx]
        +tileA[ty][19] * tileB[19][tx]
        +tileA[ty][20] * tileB[20][tx]
        +tileA[ty][21] * tileB[21][tx]
        +tileA[ty][22] * tileB[22][tx]
        +tileA[ty][23] * tileB[23][tx]
        +tileA[ty][24] * tileB[24][tx]
        +tileA[ty][25] * tileB[25][tx]
        +tileA[ty][26] * tileB[26][tx]
        +tileA[ty][27] * tileB[27][tx]
        +tileA[ty][28] * tileB[28][tx]
        +tileA[ty][29] * tileB[29][tx]
        +tileA[ty][30] * tileB[30][tx]
        +tileA[ty][31] * tileB[31][tx];

        __syncthreads ();
    }

    int Y_b = b;
    int Y_m = row;
    int Y_h = column / Width_out;
    int Y_w = column % Width_out;
    if((row < Map_out) && (column < Height_out*Width_out))
            out_4d(Y_b, Y_m, Y_h, Y_w) = acc;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}
 */

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    int inputSize = Batch * Channel * Height * Width;
    int maskSize = Map_out * Channel * K * K;
    int outputSize = Batch * Map_out * Height_out * Width_out;

    std::cout<<"Height_out "<< Height_out <<std::endl;
    std::cout<<"Width_out "<< Width_out <<std::endl;
    std::cout<<"Channel "<< Channel <<std::endl;
    std::cout<<"K "<< K <<std::endl;
    std::cout<<"Map_out "<< Map_out <<std::endl;

     __half *h_host_input, *h_host_mask, *half_device_input_ptr, *half_device_mask;

     h_host_input = (__half*) malloc(inputSize*sizeof(__half));
     for (int i=0; i<inputSize; i++)
        h_host_input[i] = __float2half(host_input[i]);

     h_host_mask = (__half*) malloc(maskSize*sizeof(__half));
     for (int i=0; i<maskSize; i++)
        h_host_mask[i] = __float2half(host_mask[i]);

    errCheck(hipMalloc((void **) &half_device_input_ptr, inputSize * sizeof(__half)));
     errCheck(hipMalloc((void **) &half_device_mask, maskSize * sizeof(__half)));
    errCheck(hipMalloc((void **) device_output_ptr, outputSize * sizeof(float)));

    errCheck(hipMemcpy(half_device_input_ptr, h_host_input, inputSize * sizeof(__half), hipMemcpyHostToDevice));
    errCheck(hipMemcpy(half_device_mask, h_host_mask, maskSize * sizeof(__half), hipMemcpyHostToDevice));


    // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;

   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   errCheck(hipEventCreate(&startWMMA));
   errCheck(hipEventCreate(&stopWMMA));
   /*
   hipblasHandle_t cublasHandle;
   cublasErrCheck(hipblasCreate(&cublasHandle));
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
    */

   printf("Running with wmma...\n");
   errCheck(hipEventRecord(startWMMA));
   conv_forward_kernel_16 <<< gridDim, blockDim >>> (*device_output_ptr, h_host_input, h_host_mask, Batch, Map_out, Channel, Height,Width, K);
   errCheck(hipEventRecord(stopWMMA));

   errCheck(hipEventDestroy(startWMMA));
   errCheck(hipEventDestroy(stopWMMA));


    //dim3 blockDim(TILE_WIDTH_16, TILE_WIDTH_16 , 1);
    //dim3 gridDim(ceil((1.0* Width_out* Height_out)/TILE_WIDTH_16), ceil((1.0*Map_out)/TILE_WIDTH_16), Batch);
    //conv_forward_kernel_16<<< gridDim, blockDim >>>(half_device_output_ptr, half_device_input_ptr, half_device_mask, Batch, Map_out, Channel, Height,Width, K );
    /*if(Map_out / 16 ){}
    else{
        dim3 blockDim(TILE_WIDTH_32, TILE_WIDTH_32 , 1);
        dim3 gridDim(ceil((1.0* Width_out* Height_out)/TILE_WIDTH_32), ceil((1.0*Map_out)/TILE_WIDTH_32), Batch);
        conv_forward_kernel_32<<< gridDim, blockDim >>>(half_device_output_ptr, half_device_input_ptr, half_device_mask, Batch, Map_out, Channel, Height,Width, K );
    }


    hipDeviceSynchronize();

    dim3 b(TILE_WIDTH_32, TILE_WIDTH_32 , 1);
    dim3 g(ceil((1.0* Width_out* Height_out)/TILE_WIDTH_32), ceil((1.0*Map_out)/TILE_WIDTH_32), Batch);
    half_to_float<<<g, b>>>(half_device_output_ptr, *device_output_ptr, Height_out*Width_out, Map_out );
    errCheck(hipMemcpy((void*) host_output, *device_output_ptr, outputSize * sizeof(float), hipMemcpyDeviceToHost));
     */


    //errCheck(hipFree(half_device_output_ptr));
    //errCheck(hipFree(half_device_input_ptr));
    //errCheck(hipFree(device_output_ptr));
    //errCheck(hipFree(half_device_mask));

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
         std::cout<<"CUDA all error: "<<hipGetErrorString(error)<<std::endl;
         exit(-1);
     }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask,
      const int Batch, const int Map_out, const int Channel, const int Height,
      const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel


    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
         std::cout<<"CUDA kern error: "<<hipGetErrorString(error)<<std::endl;
         exit(-1);
     }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    std::cout<<" Height_out: "<< Height_out <<std::endl;
    std::cout<<" Width_out: "<< Width_out <<std::endl;
    std::cout<<" Height: "<< Height <<std::endl;
    std::cout<<" Width: "<< Width <<std::endl;
     */

    // Free device memory
    //errCheck(hipFree(device_output));
    //errCheck(hipFree(device_input));
    //errCheck(hipFree(device_mask));

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
         std::cout<<"CUDA clean error: "<<hipGetErrorString(error)<<std::endl;
         exit(-1);
     }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}